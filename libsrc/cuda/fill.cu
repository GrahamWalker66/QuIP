#include "hip/hip_runtime.h"
/* Flood fill
 *
 * The basic idea is very simple; each iteration we examine each pixel;
 * it the pixel borders a filled pixel, we test it, and if the test
 * succeeds, then we fill it, also setting a global flag that something
 * has changed.  We repeat this until nothing changes.
 *
 * That implementation turned out to be very slow... One problem is
 * that unfilled pixels have to examine all of their neighbors.
 * We will try a second implementation in which when a pixel is
 * filled, it marks it's un-filled neighbors.
 *
 * No difference.  Eliminating the flag checks after each kernel
 * launch reduces the time (for 100 iterations) from 11 msec to 7 msec!
 * This could probably be speeded up quite a bit if the control
 * logic could be run on the device instead of on the host...
 *
 * But can we launch a thread array from a device function?
 * Or should we launch the whole grid and have one special thread
 * which is the master?
 * The slow implementation has one thread per pixel in the image;
 * but many iterations are required... better perhaps to have one
 * thread per filled pixel with unchecked neighbors?
 *
 * We can only synchronize threads within a block, so we would have to
 * do this with a single block.  Let's say we have one thread per
 * filled pixel...  Each pixel has up to 4 fillable neighbors (although
 * only the first seed pixel with have all 4 unfilled).  So we have
 * an array in shared memory that we fill with the pixel values. (Need
 * to check how to avoid bank conflicts!)  Then we have a table of
 * of future pixels.  Each thread gets 4 slots.  After these have
 * been filled, we would like to prune duplicates; we won't have many
 * when filling parallel to a coordinate axis, but there will be lots
 * for an oblique front...  we could use a hash function?  Or use the
 * flag image.  We could use these values:
 * 0 - unchecked
 * 1 - filled
 * 2 - queued
 * 3 - rejected
 *
 *	0 0 0 0 0    0 0 0 0 0    0 0 2 0 0
 *	0 0 0 0 0    0 0 2 0 0    0 2 1 2 0
 *	0 0 2 0 0 -> 0 2 1 2 0 -> 2 1 1 1 2
 *	0 0 0 0 0    0 0 2 0 0    0 2 1 2 0
 *	0 0 0 0 0    0 0 0 0 0    0 0 2 0 0
 *
 * Shared memory per block is only 16k, so we can't put the whole image
 * there...
 *
 * We have an array of pixels to check, sized 4 times the max number
 * of threads in a block.  We have an array of active pixels, sized
 * the max number of threads.  After syncing the threads, we need to make
 * up the new active pixel list.  We may not have enough threads to do all
 * of the pixels, so we have several lists.  After processing each list,
 * we transfer new pixels to be checked to the list, marking them as queued.
 * If we run out of space, we will have to set a flag that says we
 * have unrecorded pixels that need to be queued; if that is set when
 * we are all done, we should scan the entire image again looking for them,
 * maybe using a special flag value to indicated un-fulfilled queue request?
 * If we can allocate 2048 queue request slots it ought to be enough
 * for a 512x512 image...
 *
 * We probably want to have the shared memory allocated at launch time...
 */

#include "quip_config.h"

#ifdef HAVE_CUDA

char VersionId_cuda_fill[] = QUIP_VERSION_STRING;

#include <stdio.h>

#include <cutil.h>
#include <cutil_inline.h>

#include "my_cuda.h"
#include "cuda_supp.h"			// describe_cuda_error
#include "my_vector_functions.h"	// max_threads_per_block
#include "gpu_call_utils.h"
#include "host_call_utils.h"

// The fill routine kernel

#define FILL_IF					\
	if( fabs( dst - v ) < tol ){		\
		src1 = 1;			\
		dst = fill_val;			\
		*flag = 1;			\
		return;				\
	}

__global__ void zeroit(unsigned char* a, dim3 len )
{
	int x,y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < len.x && y < len.y ){
		a[x+y*len.y] = 0;
	}
}

__global__ void g_sp_ifl_incs(float* a, dim3 inc1,
			unsigned char* b, dim3 inc2,
			dim3 len,
			float v, float tol, float fill_val, int *flag)
{
	/* BLOCK_VARS_DECLS */
	INIT_INDICES_XY_2
	if (index1.x < len.x && index1.y < len.y ) {
		SCALE_INDICES_XY_2
		if( src1 == 0 ){	// not filled yet
			// check each neighbor if filled
			if( index2.x > 0 ){	// in-bounds
				index2.x -= inc2.x;
				if( src1 > 0 ){	// neighbor filled?
					index2.x += inc2.x;
					FILL_IF
				}
				index2.x += inc2.x;
			}
			if( index2.x < (len.x-1)*inc2.x ){
				index2.x += inc2.x;
				if( src1 > 0 ){
					index2.x -= inc2.x;
					FILL_IF
				}
				index2.x -= inc2.x;
			}
			if( index2.y > 0 ){
				index2.y -= inc2.y;
				if( src1 > 0 ){
					index2.y += inc2.y;
					FILL_IF
				}
				index2.y += inc2.y;
			}
			if( index2.y < (len.y-1)*inc2.y ){
				index2.y += inc2.y;
				if( src1 > 0 ){
					index2.y -= inc2.y;
					FILL_IF
				}
				index2.y -= inc2.y;
			}
		}
	}
}

__constant__ float test_value[1];
__constant__ float tolerance[1];
__constant__ float fill_value[1];

#define FILL_IF2					\
	if( fabs( dst - test_value[0] ) < tolerance[0] ){	\
		src1 = 1;				\
		dst = fill_value[0];			\
		return;					\
	}

__global__ void g_sp_ifl2_incs(float* a, dim3 inc1,
			unsigned char* b, dim3 inc2,
			dim3 len)
{
	/* BLOCK_VARS_DECLS */
	INIT_INDICES_XY_2
	if (index1.x < len.x && index1.y < len.y ) {
		SCALE_INDICES_XY_2
		if( src1 == 0 ){	// not filled yet
			// check each neighbor if filled
			if( index2.x > 0 ){	// in-bounds
				index2.x -= inc2.x;
				if( src1 > 0 ){	// neighbor filled?
					index2.x += inc2.x;
					FILL_IF2
				}
				index2.x += inc2.x;
			}
			if( index2.x < (len.x-1)*inc2.x ){
				index2.x += inc2.x;
				if( src1 > 0 ){
					index2.x -= inc2.x;
					FILL_IF2
				}
				index2.x -= inc2.x;
			}
			if( index2.y > 0 ){
				index2.y -= inc2.y;
				if( src1 > 0 ){
					index2.y += inc2.y;
					FILL_IF2
				}
				index2.y += inc2.y;
			}
			if( index2.y < (len.y-1)*inc2.y ){
				index2.y += inc2.y;
				if( src1 > 0 ){
					index2.y -= inc2.y;
					FILL_IF2
				}
				index2.y -= inc2.y;
			}
		}
	}
}

void h_sp_ifl( Data_Obj *dp, int x, int y, float tol, float fill_val )
{
	BLOCK_VARS_DECLS
	dim3 len, inc1, inc2;
	unsigned char *filled, b_one;
	float *f_p, v;
	int h_flag, *flag_p;
	int n_iterations;

	len.x = dp->dt_cols;
	len.y = dp->dt_rows;

	GET_MAX_THREADS(dp)
	SETUP_BLOCKS_XY

	inc1.x = dp->dt_type_inc[1];
	inc1.y = dp->dt_type_inc[2];
	inc1.z = 0;
	inc2 = inc1;

	if( hipMalloc(&flag_p,sizeof(*flag_p)) != hipSuccess ){
		NERROR1("cuda malloc error getting flag word");
	}

	/* use 2d allocator for better stride? */
	if( hipMalloc(&filled,len.x*len.y) != hipSuccess ){
		NERROR1("cuda malloc error getting filled array");
	}

	/* set filled to zero */
	CLEAR_CUDA_ERROR2("h_sp_ifl","zeroit")
	zeroit<<< NN_GPU >>>(filled,len);
	CHECK_CUDA_ERROR("h_sp_ifl","zeroit")

	// Get the value at the seed point
	f_p = (float *)dp->dt_data;
	f_p += x + y * inc1.y;

	cutilSafeCall( hipMemcpy(&v, f_p, sizeof(v),
						hipMemcpyDeviceToHost) );

	// Fill the seed point
	b_one = 1;
	cutilSafeCall( hipMemcpy(filled+x+y*len.x, &b_one, sizeof(b_one),
						hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(f_p, &fill_val, sizeof(fill_val),
						hipMemcpyHostToDevice) );


	n_iterations=0;
	do {
		/* Clear the flag */
		h_flag = 0;
		cutilSafeCall( hipMemcpy(flag_p, &h_flag, sizeof(h_flag),
						hipMemcpyHostToDevice) );

		CLEAR_CUDA_ERROR2("h_sp_ifl","g_sp_ifl_incs")
		g_sp_ifl_incs<<< NN_GPU >>>
		((float *)dp->dt_data,inc1,filled,inc2,len,v,tol,fill_val,flag_p);
		CHECK_CUDA_ERROR("h_sp_ifl","g_sp_ifl_incs")

		// download flag to see what happened.
		cutilSafeCall( hipMemcpy(&h_flag, flag_p, 1,
						hipMemcpyDeviceToHost) );
		n_iterations++;
	} while( h_flag );

	if( verbose ){
		sprintf(DEFAULT_ERROR_STRING,"Fill completed after %d iterations",n_iterations);
		advise(DEFAULT_ERROR_STRING);
	}
}

void h_sp_ifl2( Data_Obj *dp, int seed_x, int seed_y, float tol, float fill_val )
{
	BLOCK_VARS_DECLS
	dim3 len, inc1, inc2;
	unsigned char *filled, b_one;
	float *f_p, v;
	int n_iterations;

	len.x = dp->dt_cols;
	len.y = dp->dt_rows;

	GET_MAX_THREADS(dp)
	SETUP_BLOCKS_XY

	inc1.x = dp->dt_type_inc[1];
	inc1.y = dp->dt_type_inc[2];
	inc1.z = 0;
	inc2 = inc1;

	/* use 2d allocator for better stride? */
	if( hipMalloc(&filled,len.x*len.y) != hipSuccess ){
		NERROR1("cuda malloc error getting filled array");
	}

	/* set filled to zero */
	CLEAR_CUDA_ERROR2("h_sp_ifl2","zeroit")
	zeroit<<< NN_GPU >>>(filled,len);
	CHECK_CUDA_ERROR("h_sp_ifl2","zeroit")

	// Get the value at the seed point
	f_p = (float *)dp->dt_data;
	f_p += seed_x + seed_y * inc1.y;

	cutilSafeCall( hipMemcpy(&v, f_p, sizeof(v),
						hipMemcpyDeviceToHost) );

	// Fill the seed point
	b_one = 1;
	cutilSafeCall( hipMemcpy(filled+seed_x+seed_y*len.x, &b_one, sizeof(b_one),
						hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpy(f_p, &fill_val, sizeof(fill_val),
						hipMemcpyHostToDevice) );

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(fill_value), &fill_val, sizeof(float)) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(tolerance), &tol, sizeof(float)) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(test_value), &v, sizeof(float)) );

	n_iterations=0;
	for( n_iterations = 0 ; n_iterations < 300 ; n_iterations++ ){

		CLEAR_CUDA_ERROR2("h_sp_ifl2","g_sp_ifl2_incs")
		g_sp_ifl2_incs<<< NN_GPU >>>
		((float *)dp->dt_data,inc1,filled,inc2,len);
		CHECK_CUDA_ERROR("h_sp_ifl2","g_sp_ifl2_incs")

	}


	if( verbose ){
		sprintf(DEFAULT_ERROR_STRING,"Fill completed after %d iterations",n_iterations);
		advise(DEFAULT_ERROR_STRING);
	}
}


#endif /* HAVE_CUDA */

