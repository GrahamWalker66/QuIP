#include "hip/hip_runtime.h"
/* special helper function for computing centroids */

#define KERNEL_FUNC_QUALIFIER __global__

#define CENT_KERNEL( typ )						\
									\
KERNEL_FUNC_QUALIFIER void typ##_slow_cent_helper					\
( /*std_type *x_array, dim3 inc1, std_type *y_array, dim3 inc2,		\
	std_type *input, dim3 inc3, dim3 len*/ DECLARE_KERN_ARGS_SLEN_3 )	\
									\
{									\
	dim3 index;							\
	uint32_t offset1, offset2, offset3;				\
	std_type p;							\
									\
	index.x = blockIdx.x * blockDim.x + threadIdx.x;		\
	index.y = blockIdx.y * blockDim.y + threadIdx.y;		\
									\
	offset1 = index.y * inc1.x + index.x;				\
	offset2 = index.y * inc2.x + index.x;				\
	offset3 = index.y * inc3.x + index.x;				\
									\
	p = *(/*input*/ c + offset3);						\
	*(/*x_array*/a+offset1) = p * index.x;				\
	*(/*y_array*/b+offset2) = p * index.y;				\
}

#define CK( c )		CENT_KERNEL( c )

CK( type_code )

